//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2023-2024 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file geocel/vg/detail/VecgeomSetup.cu
//---------------------------------------------------------------------------//
#include "VecgeomSetup.hh"

#include <VecGeom/management/BVHManager.h>

#include "corecel/data/DeviceVector.hh"

#ifdef VECGEOM_USE_SURF
#    include <VecGeom/surfaces/cuda/BrepCudaManager.h>
#endif

#include "corecel/Assert.hh"
#include "corecel/Macros.hh"
#include "corecel/sys/KernelLauncher.device.hh"

#ifdef VECGEOM_USE_SURF
using BrepCudaManager = vgbrep::BrepCudaManager<vecgeom::Precision>;
using SurfData = vgbrep::SurfData<vecgeom::Precision>;
#endif

namespace celeritas
{
namespace detail
{
namespace
{
//---------------------------------------------------------------------------//
//! Access
struct BvhGetter
{
    vecgeom::cuda::BVH const** dest{nullptr};

    CELER_FUNCTION void operator()(ThreadId tid)
    {
        CELER_EXPECT(tid == ThreadId{0});
        *dest = vecgeom::cuda::BVHManager::GetBVH(0);
    }
};
}  // namespace

//---------------------------------------------------------------------------//
/*!
 * Get pointers to the device BVH after setup, for consistency checking.
 */
CudaPointers<vecgeom::cuda::BVH const> bvh_pointers_device()
{
    CudaPointers<vecgeom::cuda::BVH const> result;

    // Copy from kernel using 1-thread launch
    {
        DeviceVector<vecgeom::cuda::BVH const*> bvh_ptr{1, StreamId{}};
        BvhGetter execute_thread{bvh_ptr.data()};
        static KernelLauncher<decltype(execute_thread)> const launch_kernel(
            "vecgeom-get-bvhptr");
        launch_kernel(1u, StreamId{}, execute_thread);
        CELER_CUDA_CALL(hipDeviceSynchronize());
        bvh_ptr.copy_to_host({&result.kernel, 1});
    }

    // Copy from symbol using runtime API
    CELER_CUDA_CALL(hipMemcpyFromSymbol(&result.symbol,
                                         HIP_SYMBOL(vecgeom::cuda::dBVH),
                                         sizeof(vecgeom::cuda::dBVH),
                                         0,
                                         hipMemcpyDeviceToHost));
    CELER_CUDA_CALL(hipDeviceSynchronize());

    return result;
}

//---------------------------------------------------------------------------//
// VECGEOM SURFACE
//---------------------------------------------------------------------------//
#ifdef VECGEOM_USE_SURF
void setup_surface_tracking_device(SurfData const& surf_data)
{
    BrepCudaManager::Instance().TransferSurfData(surf_data);
    CELER_DEVICE_CALL_PREFIX(DeviceSynchronize());
}

void teardown_surface_tracking_device()
{
    BrepCudaManager::Instance().Cleanup();
}
#endif

//---------------------------------------------------------------------------//
}  // namespace detail
}  // namespace celeritas
